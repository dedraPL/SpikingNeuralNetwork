#include "hip/hip_runtime.h"
#include "cudaNetwork.cuh"

#include "utils.hpp"

namespace SNN 
{
	__device__ void izhikevich(float i, float& prevV, float& prevU, float a = 0.02, float dt = 0.5, float b = 0.2, float c = -65.0, float d = 8.0)
	{
        float Vans, Uans;
        if (prevV < 35)
        {
            float dv = (0.04 * prevV + 5) * prevV + 140 - prevU;
            Vans = prevV + (dv + i) * dt;
            float du = a * (b * prevV - prevU);
            Uans = prevU + dt * du;
            Vans = fminf(35, Vans);
            /*if (Vans > 35)
                Vans = 35;*/
        }
        else
        {
            Vans = c;
            Uans = prevU + d;
        }
        prevU = Uans;
        prevV = Vans;
	}

    __global__ void izhikevichLayerForward(float* V, float* U, const float* __restrict__ I, const float* __restrict__ R, const uint32_t* __restrict__ connections2DArray, const uint32_t* __restrict__ connectionsPointers, const uint32_t* __restrict__ indexes, uint32_t neuronNumber)
    {
        uint32_t index = blockIdx.x * blockDim.x + threadIdx.x;
        if (index < neuronNumber)
        {
            izhikevich(I[index], V[indexes[index]], U[indexes[index]]);
        }
    }

    __global__ void izhikevichInputLayerForward(const float* __restrict__ inputs, float* V, float* U, const uint32_t* __restrict__ indexes, uint32_t neuronNumber)
    {
        uint32_t index = blockIdx.x * blockDim.x + threadIdx.x;
        if (index < neuronNumber)
        {
            izhikevich(inputs[indexes[index]], V[indexes[index]], U[indexes[index]]);
        }
    }

    __global__ void synapse(const float* __restrict__ V, const float* __restrict__ R,
        const uint32_t* __restrict__ connections2DArray, const uint32_t* __restrict__ connectionsPointers, const uint32_t* __restrict__ indexes,
        const uint32_t* __restrict__ connectionsWIndexes, const uint32_t* __restrict__ connectionsHIndexes, uint32_t connectionsNumber,
        float* in, size_t width)
    {
        uint32_t index = (blockIdx.x * blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
        if (index < connectionsNumber)
        {
            float v_source = V[connections2DArray[connectionsPointers[indexes[connectionsWIndexes[index]]] + connectionsHIndexes[index]]];
            float v_target = V[indexes[connectionsWIndexes[index]]];
            float i = fmaxf(0, (v_source - v_target) / R[connections2DArray[connectionsPointers[indexes[connectionsWIndexes[index]]] + connectionsHIndexes[index]]]);
            in[width * connectionsHIndexes[index] + connectionsWIndexes[index]] = i;
            /*printf("index: %u in: %u target: %u source: %u\n", index, width * connectionsHIndexes[index] + connectionsWIndexes[index],
                indexes[connectionsWIndexes[index]], connections2DArray[connectionsPointers[indexes[connectionsWIndexes[index]]] + connectionsHIndexes[index]]);*/
        }
    }

    __global__ void synapse_sum(const float* __restrict__ in, float* __restrict__ out, size_t width, size_t height)
    {
        __shared__ float sdata[32][32];
        for (uint32_t w = threadIdx.x + blockDim.x * blockIdx.x; w < (width & (~((unsigned long long)(32 - 1)))) + ((width & (32 - 1)) ? 32 : 0); w += gridDim.x * blockDim.x) {          // grid-stride loop across matrix width
            sdata[threadIdx.y][threadIdx.x] = 0;
            uint32_t in_ptr = w + threadIdx.y * width;
            for (uint32_t h = threadIdx.y; h < height; h += 32) { // block-stride loop across matrix height
                sdata[threadIdx.y][threadIdx.x] += (w < width) ? in[in_ptr] : 0;
                in_ptr += width * 32;
            }
            __syncthreads();
            float my_val = sdata[threadIdx.x][threadIdx.y];
            for (int i = warpSize >> 1; i > 0; i >>= 1)                       // warp-wise parallel sum reduction
                my_val += __shfl_xor_sync(0xFFFFFFFFU, my_val, i);
            __syncthreads();
            if (threadIdx.x == 0) sdata[0][threadIdx.y] = my_val;
            __syncthreads();
            if ((threadIdx.y == 0) && ((w) < width)) out[w] = sdata[0][threadIdx.x];
        }
    }

    CUDANetwork::CUDANetwork() : nTPB{ 32, 32 }
	{
        this->V = nullptr;
        this->U = nullptr;
        this->R = nullptr;
        this->connections2DArray = nullptr;
        this->connectionsPointers = nullptr;
        this->deviceV = nullptr;
        this->deviceU = nullptr;
        this->deviceR = nullptr;
        this->deviceConnections2DArray = nullptr;
        this->deviceConnectionsPointers = nullptr;
	}

    CUDANetwork::~CUDANetwork()
    {
        freeGPUMemory();
    }

    std::vector<float> CUDANetwork::run(std::vector<float> inputs)
    {
        if (inputs.size() != inputSize)
            throw Network::InputSizeError(inputSize);

        std::vector<float> in(inputs.begin(), inputs.end());
        hipMemcpyAsync(this->deviceInputs, in.data(), in.size() * sizeof(float), hipMemcpyHostToDevice);
        propagateInput();

        copytFromGPU();
        return retrieveOutput();
    }

    std::vector<float> CUDANetwork::runContinuous(std::vector<float> inputs, uint32_t times)
    {
        if (inputs.size() != inputSize)
            throw Network::InputSizeError(inputSize);

        std::vector<float> in(inputs.begin(), inputs.end());
        hipMemcpyAsync(this->deviceInputs, in.data(), in.size() * sizeof(float), hipMemcpyHostToDevice);
        for (uint32_t i = 0; i < times; i++)
        {
            propagateInput();
        }
        
        copytFromGPU();
        return retrieveOutput();
    }

    void CUDANetwork::prepareMemory()
    {
        std::vector<float> tmpV;
        std::vector<float> tmpU;
        std::vector<float> tmpR;
        std::vector<uint32_t> tmpConnections2DArray;
        std::vector<uint32_t> tmpConnectionsPointers;
        this->outputs = new float* [this->outputSize];
        std::vector<uint32_t> outputIndexes(this->outputSize);

        for (auto const& node : this->graph)
        {
            tmpV.push_back(node.second->node->prevV);
            tmpU.push_back(node.second->node->prevU);
            tmpConnectionsPointers.push_back(tmpConnections2DArray.size());
            for (auto const& source : node.second->sources)
            {
                auto tmp = std::find_if(graph[source->dest->name]->conn.begin(), graph[source->dest->name]->conn.end(), [&](const std::shared_ptr<Synapse> s) { return s->dest->index == node.first; });
                tmpR.push_back((*tmp)->r);
                tmpConnections2DArray.push_back(source->dest->name);
            }
            if (node.second->mode == Node::NodeMode::output)
            {
                outputIndexes[node.second->index] = tmpV.size() - 1;
            }
        }

        this->connectionsIndexesNumber = new uint32_t[this->graphOrder.size() - 1]{ 0 };
        for (uint32_t i = 1; i < this->graphOrder.size(); ++i)
        {
            std::vector<uint32_t> w;
            std::vector<uint32_t> h;
            uint32_t wIndex = 0;
            for (auto const& neuronIndex : this->graphOrder[i])
            {
                uint32_t hIndex = 0;
                for (auto const& synapseIndex : graph[neuronIndex]->sources)
                {
                    w.push_back(wIndex);
                    h.push_back(hIndex);
                    connectionsIndexesNumber[i - 1]++;
                    hIndex++;
                }
                wIndex++;
            }
            this->W.push_back(w);
            this->H.push_back(h);
        }

        tmpConnectionsPointers.push_back(tmpConnections2DArray.size());
        this->V = new float[tmpV.size()];
        std::copy(tmpV.begin(), tmpV.end(), this->V);
        this->U = new float[tmpU.size()];
        std::copy(tmpU.begin(), tmpU.end(), this->U);
        this->R = new float[tmpR.size()];
        std::copy(tmpR.begin(), tmpR.end(), this->R);
        this->connections2DArray = new uint32_t[tmpConnections2DArray.size()];
        std::copy(tmpConnections2DArray.begin(), tmpConnections2DArray.end(), this->connections2DArray);
        this->connectionsPointers = new uint32_t[tmpConnectionsPointers.size()];
        std::copy(tmpConnectionsPointers.begin(), tmpConnectionsPointers.end(), this->connectionsPointers);
        this->connectionsNumber = tmpConnections2DArray.size(); 
        
        {
            uint32_t i = 0;
            for (auto const& index : outputIndexes)
            {
                this->outputs[i] = &V[index];
                i++;
            }
        }

        if (this->deviceV != nullptr)
        {
            freeGPUMemory();
        }

        hipMalloc(&this->deviceV, this->graph.size() * sizeof(float));
        hipMalloc(&this->deviceU, this->graph.size() * sizeof(float));
        hipMalloc(&this->deviceR, this->connectionsNumber * sizeof(float));
        hipMalloc(&this->deviceConnections2DArray, this->connectionsNumber * sizeof(float));
        hipMalloc(&this->deviceConnectionsPointers, (this->graph.size() + 1) * sizeof(uint32_t));

        hipMalloc(&this->deviceInputs, this->graphOrder[0].size() * sizeof(float));
        this->deviceIndexes = new uint32_t* [this->graphOrder.size()];
        this->deviceI = new float* [this->graphOrder.size() - 1];
        this->deviceIMatrix = new float* [this->graphOrder.size() - 1];
        this->deviceW = new uint32_t* [this->graphOrder.size() - 1];
        this->deviceH = new uint32_t* [this->graphOrder.size() - 1];
        for (uint32_t i = 0; i < this->graphOrder.size(); i++)
        {
            hipMalloc(&(this->deviceIndexes[i]), this->graphOrder[i].size() * sizeof(uint32_t));
            if (i > 0)
            {
                hipMalloc(&(this->deviceI[i - 1]), this->graphOrder[i].size() * sizeof(float));
                hipMalloc(&(this->deviceIMatrix[i - 1]), this->graphOrder[i - 1].size() * this->graphOrder[i].size() * sizeof(float));
                hipMalloc(&(this->deviceW[i - 1]), this->W[i - 1].size() * sizeof(uint32_t));
                hipMalloc(&(this->deviceH[i - 1]), this->H[i - 1].size() * sizeof(uint32_t));
            }
        }
        gpuErrchk();
    }

    void CUDANetwork::copyToGPU()
    {
        hipMemcpyAsync(this->deviceV, this->V, this->graph.size() * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpyAsync(this->deviceU, this->U, this->graph.size() * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpyAsync(this->deviceR, this->R, this->connectionsNumber * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpyAsync(this->deviceConnections2DArray, this->connections2DArray, this->connectionsNumber * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpyAsync(this->deviceConnectionsPointers, this->connectionsPointers, (this->graph.size() + 1) * sizeof(uint32_t), hipMemcpyHostToDevice);

        for (uint32_t i = 0; i < this->graphOrder.size(); i++)
        {
            hipMemcpyAsync(this->deviceIndexes[i], this->graphOrder[i].data(), this->graphOrder[i].size() * sizeof(uint32_t), hipMemcpyHostToDevice);
            if (i > 0)
            {
                hipMemsetAsync(this->deviceIMatrix[i - 1], 0, this->graphOrder[i - 1].size() * this->graphOrder[i].size() * sizeof(float));
                hipMemcpyAsync(this->deviceW[i - 1], this->W[i - 1].data(), this->W[i - 1].size() * sizeof(uint32_t), hipMemcpyHostToDevice);
                hipMemcpyAsync(this->deviceH[i - 1], this->H[i - 1].data(), this->H[i - 1].size() * sizeof(uint32_t), hipMemcpyHostToDevice);
            }
        }

        hipDeviceSynchronize();
        gpuErrchk();
    }

    void CUDANetwork::copytFromGPU()
    {
        hipMemcpyAsync(this->V, this->deviceV, this->graph.size() * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpyAsync(this->U, this->deviceU, this->graph.size() * sizeof(float), hipMemcpyDeviceToHost);
        //hipMemcpy(this->R, this->deviceR, this->connectionsNumber * sizeof(float), hipMemcpyDeviceToHost);
        //hipMemcpy(this->connections2DArray, this->deviceConnections2DArray, this->connectionsNumber * sizeof(float), hipMemcpyDeviceToHost);
        //hipMemcpy(this->connectionsPointers, this->deviceConnectionsPointers, (this->graph.size() + 1) * sizeof(uint32_t), hipMemcpyDeviceToHost);
        //hipDeviceSynchronize();
        gpuErrchk();
    }

    void CUDANetwork::freeGPUMemory()
    {
        hipFree(&this->deviceV);
        hipFree(&this->deviceU);
        hipFree(&this->deviceR);
        hipFree(&this->deviceConnections2DArray);
        hipFree(&this->deviceConnectionsPointers);

        hipFree(&this->deviceInputs);
        for (uint32_t i = 0; i < this->graph.size(); i++)
        {
            hipFree(&this->deviceIndexes[i]);
            if (i > 0)
            {
                hipFree(&this->deviceI[i - 1]);
                hipFree(&this->deviceIMatrix[i - 1]);
                hipFree(&this->deviceW[i - 1]);
                hipFree(&this->deviceH[i - 1]);
            }
        }
        hipFree(&this->deviceIndexes);
        hipFree(&this->deviceI);
        hipFree(&this->deviceIMatrix);
        hipFree(&this->deviceW);
        hipFree(&this->deviceH);
        hipGetLastError();
    }
    
    std::vector<float> CUDANetwork::retrieveOutput()
    {
        std::vector<float> out(this->outputSize);
        for (uint32_t i = 0; i < outputSize; i++)
        {
            memcpy(&out[i], this->outputs[i], sizeof(float));
        }
        return out;
    }

    void CUDANetwork::propagateInput()
    {
        dim3 threads(1024);

        uint32_t layerNumber = 0;
        for (auto const& layer : this->graphOrder)
        {
            dim3 blocksIz((layer.size() + threads.x - 1) / threads.x);
            if (layerNumber == 0)
            {   
                izhikevichInputLayerForward KERNEL2(blocksIz, threads)(this->deviceInputs, this->deviceV, this->deviceU, this->deviceIndexes[layerNumber], layer.size());
            }
            else
            {
                dim3 blocksSyn((connectionsIndexesNumber[layerNumber - 1] + threads.x - 1) / threads.x);
                synapse KERNEL2(blocksSyn, threads)(this->deviceV, this->deviceR,
                    this->deviceConnections2DArray, this->deviceConnectionsPointers, this->deviceIndexes[layerNumber],
                    this->deviceW[layerNumber - 1], this->deviceH[layerNumber - 1], this->connectionsIndexesNumber[layerNumber - 1],
                    this->deviceIMatrix[layerNumber - 1], layer.size());
                synapse_sum KERNEL2((layer.size() + this->block_x - 1) / this->block_x, nTPB)(this->deviceIMatrix[layerNumber - 1], this->deviceI[layerNumber - 1], layer.size(), graphOrder[layerNumber - 1].size());
                gpuErrchk();
                izhikevichLayerForward KERNEL2(blocksIz, threads)(this->deviceV, this->deviceU, this->deviceI[layerNumber - 1], this->deviceR, this->deviceConnections2DArray, this->deviceConnectionsPointers, this->deviceIndexes[layerNumber], layer.size());
            }
            gpuErrchk();

            layerNumber++;
        }
    }
}